#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <windows.h>
#include <sm_20_atomic_functions.h>
#include <hip/hip_runtime.h>
#include "BigNum.h"

/*
λ���ı䣬�������䡣 
*/

/*
*****************************************************************************************************************************************************
1.CUDA DO NOT support using kernel function in the class
2.Transfering data from GPU to CPU will take much time
3.kernel function CAN NOT invoking function which in HOST
4.kernel function can ONLY define and declare in kernel.cu(main function)
*****************************************************************************************************************************************************
*/

//Need to optimize
__global__ void mul(dword *a, dword *b, qword *c, int aLen, int bLen)
{
	__shared__ qword x[1024];
	int tId = threadIdx.x + blockIdx.x * blockDim.x;//1���еĴ��
	int xp = threadIdx.x;//�����λ�ڵڼ�С��

	x[xp] = 0;

	for(int i = 0; i < aLen; i++)
	{
		int temp_p = tId - i;

		if(temp_p >= 0 && temp_p < bLen)
			x[xp] += (qword)a[MAXLEN - 1 - i] * b[MAXLEN - 1 - temp_p]; 
		else if(temp_p < 0)
			break;
	}

	c[MAXLEN -1 - tId] = x[xp];
}

void mulZero(BigNum &a, BigNum &b, BigNum *c)
{
	c->signnum = 0;
}

void BigNummul(BigNum &a, BigNum &b,BigNum *c, int signnum)
{
	FILE *fp=fopen("datamul.txt","a+");
	LARGE_INTEGER freq;  
	LARGE_INTEGER start_t, stop_t;  
	double exe_time;  
	hipEvent_t start,stop;
	float elapsedtime = 0;

	qword *cc;
	qword *cValue;
	dword *aValue, *bValue;

	int len = a.dwLen + b.dwLen;
	int THREAD = len >= MAXT ? MAXT : len;
	int BLOCK = len >= MAXT ? len / MAXT + 1: 1;
	//int BLOCK = len >= MAXT ? (len + THREAD - 1) / THREAD: 1;

	if(signnum)
		c->signnum = -1;
	else
		c->signnum = 1;
	c->dwLen = a.dwLen + b.dwLen;
	c->start = MAXLEN - c->dwLen;

	cc = (qword *)malloc(MAXLEN * sizeof(qword));
	hipMalloc((void **)&aValue, MAXLEN * sizeof(dword));
	hipMalloc((void **)&bValue, MAXLEN * sizeof(dword));
	hipMalloc((void **)&cValue, MAXLEN * sizeof(qword));
	
	hipMemcpy(aValue, a.value, MAXLEN * sizeof(dword), hipMemcpyHostToDevice);
	hipMemcpy(bValue, b.value, MAXLEN * sizeof(dword), hipMemcpyHostToDevice);
	hipMemset(cValue,0,MAXLEN * sizeof(qword));
	hipEventCreate(&start);//gpu time
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	mul<<<BLOCK, THREAD>>>(aValue, bValue, cValue, a.dwLen, b.dwLen);
	//hipMemcpy(cc, cValue, MAXLEN * sizeof(qword), hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(start); 
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	fprintf(fp,"����ʱ�䣺%.3f ms\n",elapsedtime);
	
	hipEventCreate(&start);//gpu time
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	hipMemcpy(cc, cValue, MAXLEN * sizeof(qword), hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);
	hipEventSynchronize(start); 
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	//fprintf(fp,"����ʱ�䣺%.3f ms\n",elapsedtime);
	
	QueryPerformanceFrequency(&freq);  
	QueryPerformanceCounter(&start_t);//cpu time 

	for(int i = 0; i < a.dwLen+b.dwLen; i++)
	{
		cc[MAXLEN - 1 - i - 1] += cc[MAXLEN - 1 - i] / DIGIT; 
		cc[MAXLEN - 1 - i] %= DIGIT;
	}

	QueryPerformanceCounter(&stop_t);  
	exe_time = 1e3*(stop_t.QuadPart-start_t.QuadPart)/freq.QuadPart;  
	//fprintf(fp,"ʱ�䣺%.3f ms\n",exe_time+elapsedtime);

	//useless management
	for(int i = MAXLEN - 1; i >= c->start; i--)
		c->value[i] = cc[i];

	if(!c->value[MAXLEN - (a.dwLen + b.dwLen)])
	{
		c->start++;
		c->dwLen--;
	}

	if(c->start <= 0)
		c->start = 0;

	fclose(fp);
	hipFree(aValue);
	hipFree(bValue);
	hipFree(cValue);
}

int main(void)
{
	BigNum a, b, c;
	char oper;
	//char temp[MAXLEN];

	printf("Input two value(DO NOT Larger than 10000 digit)\n");

	a.make();
	b.make();
	getchar();

	/*gets(temp);
	a.getBuf(temp, strlen(temp));
	a.toDword(temp, strlen(temp));

	gets(temp);
	b.getBuf(temp, strlen(temp));
	b.toDword(temp, strlen(temp));*/

	printf("Choose one: + - * /\n");
	oper=getchar();

	switch(oper)
	{
		/*
	case'+':
		if(a.isZero() || b.isZero())
			addZero(a, b, &c);
		else if(a.signnum == b.signnum)
			BigNumadd(a, b, &c);
		else
			BigNumsub(a, b, &c);
		break;
	case'-': 
		if(a.isZero() || b.isZero())
			subZero(a, b, &c);
		else if(a.signnum != b.signnum)
			BigNumadd(a, b, &c);
		else
			BigNumsub(a, b, &c);
		break;
		*/
	case'*': 
		if(a.isZero() || b.isZero())
			mulZero(a, b, &c);
		else if(a.signnum != b.signnum)
			BigNummul(a, b, &c, 1);
		else 
			BigNummul(a, b, &c, 0);
		break;
	/*case'/': BigNumdiv(a, b, &c);	break;*/
	}

	//c.print();
	//c.fprint();

	return 0;
}